#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "utils.h"

#define SIZE	1e6
#define THREADS	256
#define BLOCKS MIN(32, (SIZE + THREADS - 1)/ THREADS)

#define PI		3.14159265
#define RECTS 1000000 //1e6
#define START 0.0
#define END 	PI

__global__ void integration(double *x, double *dx, double *results) {
	__shared__ double cache[THREADS];

	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	int cacheIndex = threadIdx.x;

	double acum = 0;
	while (tid < SIZE) {
    acum += sin( (*x) + (tid * (*dx)) );
		tid += blockDim.x * gridDim.x;
	}

	cache[cacheIndex] = acum;

	__syncthreads();

	int i = blockDim.x / 2;
	while (i > 0) {
		if (cacheIndex < i) {
			cache[cacheIndex] += cache[cacheIndex + i];
		}
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0) {
		results[blockIdx.x] = cache[cacheIndex];
	}
}

int main(int argc, char* argv[]) {
	double x, dx, *results;
  double *d_x, *d_dx, *d_r;
	double ms;
  int i;

  x = START;
  dx = (END - START) / RECTS;

	results = (double*) malloc( BLOCKS * sizeof(double) );

	hipMalloc( (void**) &d_x, sizeof(double));
  hipMalloc( (void**) &d_dx, sizeof(double));
	hipMalloc( (void**) &d_r, BLOCKS * sizeof(double) );

	hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_dx, &dx, sizeof(double), hipMemcpyHostToDevice);

	printf("Starting...\n");
	ms = 0;
	for (i = 1; i <= N; i++) {
		start_timer();

		integration<<<BLOCKS, THREADS>>> (d_x, d_dx, d_r);

		ms += stop_timer();
	}

	hipMemcpy(results, d_r, BLOCKS * sizeof(double), hipMemcpyDeviceToHost);

	double acum = 0;
	for (i = 0; i < BLOCKS; i++) {
		acum += results[i];
	}

	printf("area = %.5lf\n", (acum * dx));
	printf("avg time = %.5lf\n", (ms / N));

  hipFree(d_x);
  hipFree(d_dx);
	hipFree(d_r);

	free(results);
	return 0;
}
