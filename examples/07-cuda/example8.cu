// =================================================================
//
// File: example8.cu
// Author(s):
// Description: This file contains the code that implements the
//				enumeration sort algorithm using CUDA.
//
// Copyright (c) 2020 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define SIZE 10000
#define THREADS 256
#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))
