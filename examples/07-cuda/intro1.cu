// =================================================================
//
// File: intro1.cu
// Author: Pedro Perez
// Description: This file shows some of the basic CUDA directives.
//
// Copyright (c) 2020 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
}

int main(int argc, char* argv[]) {
	int a, b, c;
	int *d_a, *d_b, *d_c;

	hipMalloc((void**) &d_a, sizeof(int));
	hipMalloc((void**) &d_b, sizeof(int));
	hipMalloc((void**) &d_c, sizeof(int));

	scanf("%i %i", &a, &b);

	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

	add<<<1, 1>>>(d_a, d_b, d_c);

	hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

	printf("c = %i\n", c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
