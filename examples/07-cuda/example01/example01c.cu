#include "hip/hip_runtime.h"
// =================================================================
//
// File: example01c.cpp
// Author: Pedro Perez
// Description: This file implements the addition of two vectors
//				using CUDA technology. In this case, we use 
//				a ''matrix'' of cores.
//
// Copyright (c) 2023 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <iostream>
#include <iomanip>
#include <chrono>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "utils.h"

using namespace std;
using namespace std::chrono;

#define SIZE 10000000 // 1e7
#define THREADS 512
#define BLOCKS	max(32, ((SIZE / THREADS) + 1))

__global__ void add(int *a, int *b, int *c) {
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	if (i < SIZE) {
		c[i] = a[i] + b[i];
	}
}

int main(int argc, char* argv[]) {
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;

	// These variables are used to keep track of the execution time.
	high_resolution_clock::time_point start, end;
	double timeElapsed;

	a = new int [SIZE];
	b = new int [SIZE];
	c = new int [SIZE];

	fill_array(a, SIZE);
	display_array("a:", a);
	fill_array(b, SIZE);
	display_array("b:", b);

	hipMalloc((void**) &d_a, SIZE * sizeof(int));
	hipMalloc((void**) &d_b, SIZE * sizeof(int));
	hipMalloc((void**) &d_c, SIZE * sizeof(int));

	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);

	cout << "Starting...\n";
	timeElapsed = 0;
	for (int j = 0; j < N; j++) {
		start = high_resolution_clock::now();

		add<<<BLOCKS, THREADS>>>(d_a, d_b, d_c);

		end = high_resolution_clock::now();
		timeElapsed += 
			duration<double, std::milli>(end - start).count();
	}

	hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	display_array("c:", c);
	cout << "avg time = " << fixed << setprecision(3) 
		 << (timeElapsed / N) <<  " ms\n";

	hipFree(d_c);
	hipFree(d_b);
	hipFree(d_a);

	delete [] a;
	delete [] b;
	delete [] c;

	return 0;
}
