// =================================================================
//
// File: intro2.cu
// Author: Pedro Perez
// Description: This file shows some of the basic CUDA directives.
//
// Copyright (c) 2020 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ float fx(float x, float y) {
	return x + y;
}

__global__ void kernel(void) {
	printf("res = %f\n", fx(1.0, 2.0));
}

int main(int argc, char* argv[]) {
	kernel<<<1, 1>>>();
	hipDeviceSynchronize();

	return 0;
}
