#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ float fx(float x, float y) {
	return x + y;
}

__global__ void kernel(void) {
	printf("res = %f\n", fx(1.0, 2.0));
}

int main(int argc, char* argv[]) {
	kernel<<<1, 1>>>();
	hipDeviceSynchronize();

	return 0;
}
