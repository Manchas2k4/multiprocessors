#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel(void) {
	printf("GPU: Hello world\n");
}

int main(int argc, char* argv[]) {
	kernel<<<2, 4>>>();
	hipDeviceSynchronize();

	return 0;
}
