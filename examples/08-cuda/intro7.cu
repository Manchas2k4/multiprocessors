
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char* argv[]) {
	int i, count;
	hipDeviceProp_t prop;
	
	hipGetDeviceCount(&count);
	for (i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("Device name: %s\n", prop.name);
	}
	return 0;
}
