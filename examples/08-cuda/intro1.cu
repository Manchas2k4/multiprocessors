#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
}

int main(int argc, char* argv[]) {
	int a, b, c;
	int *d_a, *d_b, *d_c;
	
	hipMalloc((void**) &d_a, sizeof(int));
	hipMalloc((void**) &d_b, sizeof(int));
	hipMalloc((void**) &d_c, sizeof(int));
	
	scanf("%i %i", &a, &b);
	
	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
	
	add<<<1, 1>>>(d_a, d_b, d_c);
	
	hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	
	printf("c = %i\n", c);
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	return 0;
}
